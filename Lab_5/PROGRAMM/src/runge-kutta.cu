#include "hip/hip_runtime.h"
#include "../include/N-body-problem-cuda.cuh"


/**
 * @brief Метод Рунге-Кутта для решения задачи о движении N тел с использованием CUDA.
 * @param path Путь к файлу для вывода данных.
 * @param global_m Массы тел.
 * @param global_r Начальные координаты тел.
 * @param global_v Начальные скорости тел.
 * @param tau Шаг интегрирования.
 * @param T Время интегрирования.
 * @param output Флаг записи результата.
 * @return Среднее время выполнения одного шага.
 */
float Runge_Kutta(const std::string& path, const std::vector<mytype>& global_m,
                  std::vector<mytype>& global_r, std::vector<mytype>& global_v,
                  mytype tau, mytype T, bool output) {

    int N = global_m.size(), N3 = 3 * N;
    mytype *device_m = nullptr;
    mytype *device_r = nullptr;
    mytype *device_v = nullptr;
    mytype *kr1 = nullptr;
    mytype *kv1 = nullptr;
    mytype *kr2 = nullptr;
    mytype *kv2 = nullptr;
    mytype *kr3 = nullptr;
    mytype *kv3 = nullptr;
    mytype *kr4 = nullptr;
    mytype *kv4 = nullptr;
    mytype *temp_device_r = nullptr;
    mytype *temp_device_v = nullptr;
    mytype tau2 = tau / 2, t0 = 0.0;
    dim3 blocks((N + BS - 1) / BS), threads(BS);

    // Вывод числа тел
    std::cout << "[LOG]: N = " << N << std::endl;

    if (output) {
        for (size_t i = 0; i < N; ++i) {
            write(path, {global_r[3 * i], global_r[3 * i + 1], global_r[3 * i + 2]}, t0, i + 1);
        }
    }

    // Выделение памяти на устройстве
    hipMalloc(&device_m, N * sizeof(mytype));
    hipMalloc(&device_r, N3 * sizeof(mytype));
    hipMalloc(&device_v, N3 * sizeof(mytype));
    hipMalloc(&temp_device_r, N3 * sizeof(mytype));
    hipMalloc(&temp_device_v, N3 * sizeof(mytype));
    hipMalloc(&kr1, N3 * sizeof(mytype));
    hipMalloc(&kr2, N3 * sizeof(mytype));
    hipMalloc(&kr3, N3 * sizeof(mytype));
    hipMalloc(&kr4, N3 * sizeof(mytype));
    hipMalloc(&kv1, N3 * sizeof(mytype));
    hipMalloc(&kv2, N3 * sizeof(mytype));
    hipMalloc(&kv3, N3 * sizeof(mytype));
    hipMalloc(&kv4, N3 * sizeof(mytype));

    // Копирование данных на устройство
    hipMemcpy(device_m, global_m.data(), N * sizeof(mytype), hipMemcpyHostToDevice);
    hipMemcpy(device_r, global_r.data(), N3 * sizeof(mytype), hipMemcpyHostToDevice);
    hipMemcpy(device_v, global_v.data(), N3 * sizeof(mytype), hipMemcpyHostToDevice);

    hipEvent_t start, finish;
    hipEventCreate(&start);
    hipEventCreate(&finish);

    float time = 0.0f;
    hipEventRecord(start);

    int iter = 0;
    while (t0 <= T) {
        // Расчёт этапов метода Рунге-Кутта
        f<<<blocks, threads>>>(kr1, kv1, device_m, device_r, device_v, N);

        add<<<blocks, threads>>>(device_r, kr1, tau2, temp_device_r, N);
        add<<<blocks, threads>>>(device_v, kv1, tau2, temp_device_v, N);
        f<<<blocks, threads>>>(kr2, kv2, device_m, temp_device_r, temp_device_v, N);

        add<<<blocks, threads>>>(device_r, kr2, tau2, temp_device_r, N);
        add<<<blocks, threads>>>(device_v, kv2, tau2, temp_device_v, N);
        f<<<blocks, threads>>>(kr3, kv3, device_m, temp_device_r, temp_device_v, N);

        add<<<blocks, threads>>>(device_r, kr3, tau, temp_device_r, N);
        add<<<blocks, threads>>>(device_v, kv3, tau, temp_device_v, N);
        f<<<blocks, threads>>>(kr4, kv4, device_m, temp_device_r, temp_device_v, N);

        summarize<<<blocks, threads>>>(device_r, device_v, tau, kr1, kv1, kr2, kv2, kr3, kv3, kr4, kv4, N);

        t0 += tau;
        ++iter;

        if (output && ((int) (round(1000 * t0)) % 100 == 0)) {
            hipMemcpy(global_r.data(), device_r, N3 * sizeof(mytype), hipMemcpyDeviceToHost);
            for (size_t i = 0; i < N; ++i) {
                write(path, {global_r[3 * i], global_r[3 * i + 1], global_r[3 * i + 2]}, t0, i + 1);
            }
        }
    }

    hipDeviceSynchronize();
    hipEventRecord(finish);
    hipEventSynchronize(finish);

    hipEventElapsedTime(&time, start, finish);

    // Копирование данных обратно на хост
    hipMemcpy(global_r.data(), device_r, N3 * sizeof(mytype), hipMemcpyDeviceToHost);
    hipMemcpy(global_v.data(), device_v, N3 * sizeof(mytype), hipMemcpyDeviceToHost);

    // Освобождение памяти
    hipFree(device_m);
    hipFree(device_r);
    hipFree(device_v);
    hipFree(temp_device_r);
    hipFree(temp_device_v);
    hipFree(kr1);
    hipFree(kr2);
    hipFree(kr3);
    hipFree(kr4);
    hipFree(kv1);
    hipFree(kv2);
    hipFree(kv3);
    hipFree(kv4);

    return time / iter;
}