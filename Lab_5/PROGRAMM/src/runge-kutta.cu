#include "hip/hip_runtime.h"
#include "../include/N-body-problem-cuda.cuh"




/**
 * @brief Метод Рунге-Кутта для решения задачи о движении N тел с использованием CUDA.
 * @param path Путь к файлу для вывода данных.
 * @param global_m Массы тел.
 * @param global_r Начальные координаты тел.
 * @param global_v Начальные скорости тел.
 * @param tau Шаг интегрирования.
 * @param T Время интегрирования.
 * @param output Флаг записи результата.
 * @return Среднее время выполнения одного шага.
 */
float Runge_Kutta(const std::string& path, const std::vector<mytype>& global_m,
                  std::vector<mytype3>& global_r, std::vector<mytype3>& global_v,
                  mytype tau, mytype T, bool output) {

    int N = global_m.size();  // Количество тел

    mytype3 *device_r = nullptr, *device_v = nullptr;
    mytype3 *kr1 = nullptr, *kv1 = nullptr;
    mytype3 *kr2 = nullptr, *kv2 = nullptr;
    mytype3 *kr3 = nullptr, *kv3 = nullptr;
    mytype3 *kr4 = nullptr, *kv4 = nullptr;
    mytype3 *temp_device_r = nullptr, *temp_device_v = nullptr;
    mytype *device_m = nullptr;

    mytype tau2 = tau / 2, t0 = 0.0;

    dim3 blocks((N + BS - 1) / BS); // Число блоков
    dim3 threads(BS);               // Число потоков

    std::cout << "[LOG]: N = " << N << std::endl;

    // Запись начального положения
    if (output) {
        for (size_t i = 0; i < N; ++i) {
            write(path, global_r[i], t0, i + 1);
        }
    }

    // Выделение памяти на устройстве
    hipMalloc(&device_m, N * sizeof(mytype));
    hipMalloc(&device_r, N * sizeof(mytype3));
    hipMalloc(&device_v, N * sizeof(mytype3));
    hipMalloc(&temp_device_r, N * sizeof(mytype3));
    hipMalloc(&temp_device_v, N * sizeof(mytype3));
    hipMalloc(&kr1, N * sizeof(mytype3));
    hipMalloc(&kr2, N * sizeof(mytype3));
    hipMalloc(&kr3, N * sizeof(mytype3));
    hipMalloc(&kr4, N * sizeof(mytype3));
    hipMalloc(&kv1, N * sizeof(mytype3));
    hipMalloc(&kv2, N * sizeof(mytype3));
    hipMalloc(&kv3, N * sizeof(mytype3));
    hipMalloc(&kv4, N * sizeof(mytype3));

    // Копирование данных на устройство
    hipMemcpy(device_m, global_m.data(), N * sizeof(mytype), hipMemcpyHostToDevice);
    hipMemcpy(device_r, global_r.data(), N * sizeof(mytype3), hipMemcpyHostToDevice);
    hipMemcpy(device_v, global_v.data(), N * sizeof(mytype3), hipMemcpyHostToDevice);

    hipEvent_t start, finish;
    hipEventCreate(&start);
    hipEventCreate(&finish);

    float time = 0.0f;
    hipEventRecord(start);

    int iter = 0;
    while (t0 <= T) {
        // Расчёт этапов метода Рунге-Кутта
        f<<<blocks, threads>>>(kr1, kv1, device_m, device_r, device_v, N);

        add<<<blocks, threads>>>(device_r, kr1, tau2, temp_device_r, N);
        add<<<blocks, threads>>>(device_v, kv1, tau2, temp_device_v, N);
        f<<<blocks, threads>>>(kr2, kv2, device_m, temp_device_r, temp_device_v, N);

        add<<<blocks, threads>>>(device_r, kr2, tau2, temp_device_r, N);
        add<<<blocks, threads>>>(device_v, kv2, tau2, temp_device_v, N);
        f<<<blocks, threads>>>(kr3, kv3, device_m, temp_device_r, temp_device_v, N);

        add<<<blocks, threads>>>(device_r, kr3, tau, temp_device_r, N);
        add<<<blocks, threads>>>(device_v, kv3, tau, temp_device_v, N);
        f<<<blocks, threads>>>(kr4, kv4, device_m, temp_device_r, temp_device_v, N);

        summarize<<<blocks, threads>>>(device_r, device_v, tau, kr1, kv1, kr2, kv2, kr3, kv3, kr4, kv4, N);

        t0 += tau;
        ++iter;

        if (output) {
            hipMemcpy(global_r.data(), device_r, N * sizeof(float3), hipMemcpyDeviceToHost);
            for (size_t i = 0; i < N; ++i) {
                write(path, global_r[i], t0, i + 1);
            }
        }
    }

    hipEventRecord(finish);
    hipEventSynchronize(finish);

    hipEventElapsedTime(&time, start, finish);

    // Копирование данных обратно на хост
    hipMemcpy(global_r.data(), device_r, N * sizeof(mytype3), hipMemcpyDeviceToHost);
    hipMemcpy(global_v.data(), device_v, N * sizeof(mytype3), hipMemcpyDeviceToHost);

    // Освобождение памяти
    hipFree(device_m);
    hipFree(device_r);
    hipFree(device_v);
    hipFree(temp_device_r);
    hipFree(temp_device_v);
    hipFree(kr1);
    hipFree(kr2);
    hipFree(kr3);
    hipFree(kr4);
    hipFree(kv1);
    hipFree(kv2);
    hipFree(kv3);
    hipFree(kv4);

    return time / iter;
}
